#include "hip/hip_runtime.h"
#include "spring_model_kernel.cuh"


__global__
void cuda_particles_kernel(sParticle *particles, t_num dt, unsigned int size)
{
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < size)
    {
      t_num Fx = 0.0;
      t_num Fy = 0.0;
      t_num Fz = 0.0;

      t_num ks = 2.0;
      t_num kf = 0.1;

      t_num fsx = 0.0;
      t_num fsy = 0.0;
      t_num fsz = 0.0;

      sParticle particle = particles[idx];

      for (unsigned int j = 0; j < INTERACTION_POINT_MAX; j++)
      {
        int idx_b = particle.interaction_idx[j];
        if (idx_b >= 0)
        {
          t_num dx_new = particle.x - particles[idx_b].x;
          t_num dy_new = particle.y - particles[idx_b].y;
          t_num dz_new = particle.z - particles[idx_b].z;

          fsx+= dx_new - particle.interaction_idx_dx[j];
          fsy+= dy_new - particle.interaction_idx_dy[j];
          fsz+= dz_new - particle.interaction_idx_dz[j];
        }
      }

      fsx = -fsx*ks;
      fsy = -fsy*ks;
      fsz = -fsz*ks;

      t_num ffx = -kf*particle.vx;
      t_num ffy = -kf*particle.vy;
      t_num ffz = -kf*particle.vz;

      particle.fx = Fx + fsx + ffx;
      particle.fy = Fy + fsy + ffy;
      particle.fz = Fz + fsz + ffz;

      particle.ax = particle.fx/particle.mass;
      particle.ay = particle.fy/particle.mass;
      particle.az = particle.fz/particle.mass;

      particle.vx+= particle.ax*dt;
      particle.vy+= particle.ay*dt;
      particle.vz+= particle.az*dt;

      particle.x+= particle.vx*dt;
      particle.y+= particle.vy*dt;
      particle.z+= particle.vz*dt;

      particles[idx] = particle;
    }
}

__host__
void cpu_particles_kernel(sParticle *particles, t_num dt, unsigned int size)
{
    for (unsigned int idx = 0; idx < size; idx++)
    {
      t_num Fx = 0.0;
      t_num Fy = 0.0;
      t_num Fz = 0.0;

      t_num ks = 1.0;
      t_num kf = 0.1;

      t_num fsx = 0.0;
      t_num fsy = 0.0;
      t_num fsz = 0.0;

      for (unsigned int j = 0; j < INTERACTION_POINT_MAX; j++)
      {
        int idx_b = particles[idx].interaction_idx[j];
        if (idx_b >= 0)
        {
          t_num dx_new = particles[idx].x - particles[idx_b].x;
          t_num dy_new = particles[idx].y - particles[idx_b].y;
          t_num dz_new = particles[idx].z - particles[idx_b].z;

          fsx+= dx_new - particles[idx].interaction_idx_dx[j];
          fsy+= dy_new - particles[idx].interaction_idx_dy[j];
          fsz+= dz_new - particles[idx].interaction_idx_dz[j];
        }
      }

      fsx = -fsx*ks;
      fsy = -fsy*ks;
      fsz = -fsz*ks;

      t_num ffx = -kf*particles[idx].vx;
      t_num ffy = -kf*particles[idx].vy;
      t_num ffz = -kf*particles[idx].vz;

      particles[idx].fx = Fx + fsx + ffx;
      particles[idx].fy = Fy + fsy + ffy;
      particles[idx].fz = Fz + fsz + ffz;

      particles[idx].ax = particles[idx].fx/particles[idx].mass;
      particles[idx].ay = particles[idx].fy/particles[idx].mass;
      particles[idx].az = particles[idx].fz/particles[idx].mass;

      particles[idx].vx+= particles[idx].ax*dt;
      particles[idx].vy+= particles[idx].ay*dt;
      particles[idx].vz+= particles[idx].az*dt;

      particles[idx].x+= particles[idx].vx*dt;
      particles[idx].y+= particles[idx].vy*dt;
      particles[idx].z+= particles[idx].vz*dt;
    }
}

void particles_kernel(sParticle *particles, t_num dt, unsigned int size)
{
  #ifdef _PARTICLES_USE_CUDA_
    dim3 block(32);
    dim3 grid((size + block.x - 1)/block.x);

    cuda_particles_kernel<<<grid, block>>>(particles, dt, size);
    hipDeviceSynchronize();
  #else
    cpu_particles_kernel(particles, dt, size);
  #endif
}
