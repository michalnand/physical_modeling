#include "cuda_particle_allocator.cuh"
#include <iostream>

CudaParticleAllocator cuda_particle_allocator;


CudaParticleAllocator::CudaParticleAllocator()
{

}

CudaParticleAllocator::~CudaParticleAllocator()
{

}

sParticle* CudaParticleAllocator::malloc(unsigned int count)
{
  mutex.lock();

  sParticle *result = nullptr;

  hipMalloc(&result, count*sizeof(sParticle));

  mutex.unlock();

  clear(result, count);

  return result;
}

void CudaParticleAllocator::free(void *ptr)
{
  mutex.lock();

  if (ptr != nullptr)
  {
    hipFree(ptr);
    ptr = nullptr;
  }

  mutex.unlock();
}

void CudaParticleAllocator::host_to_device(sParticle *dev_ptr, sParticle *host_ptr, unsigned int size)
{
  hipMemcpy(dev_ptr, host_ptr, size*sizeof(sParticle), hipMemcpyHostToDevice);
}

void CudaParticleAllocator::device_to_host(sParticle *host_ptr, sParticle *dev_ptr, unsigned int size)
{
  hipMemcpy(host_ptr, dev_ptr, size*sizeof(sParticle), hipMemcpyDeviceToHost);
}

void CudaParticleAllocator::device_to_device(sParticle *dest_ptr, sParticle *src_ptr, unsigned int size)
{
  hipMemcpy(dest_ptr, src_ptr, size*sizeof(sParticle), hipMemcpyDeviceToDevice);
}

void CudaParticleAllocator::clear(sParticle *result, unsigned int size)
{
  hipMemset(result, 0, size*sizeof(sParticle));
}
