#include "spring_model_kernel.cuh"

#include <stdio.h>

sParticle* cuda_spring_model_allocate(unsigned int count)
{
  sParticle *result = nullptr;
  auto res = hipMalloc(&result, count*sizeof(sParticle));

  return result;
}


void cuda_spring_model_init(sParticle *cu_particles, std::vector<sParticle> &particles)
{
  cu_particles = cuda_spring_model_allocate(particles.size());

  auto res = hipMemcpy(cu_particles, &particles[0], particles.size()*sizeof(sParticle), hipMemcpyHostToDevice);
}

void cuda_spring_model_to_host(std::vector<sParticle> &particles, sParticle *cu_particles)
{
  hipMemcpy(&particles[0], cu_particles, particles.size()*sizeof(sParticle), hipMemcpyDeviceToHost);
} 

void cuda_spring_model_compute(sParticle *cu_particles)
{

}
